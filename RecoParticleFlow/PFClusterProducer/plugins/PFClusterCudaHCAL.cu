#include "hip/hip_runtime.h"

#include <cmath>
#include <iostream>

// CUDA include files
#include <hip/hip_runtime.h>

// CMSSW include files
#include "HeterogeneousCore/CUDAUtilities/interface/cudaCheck.h"
#include "RecoParticleFlow/PFClusterProducer/plugins/PFClusterCudaHCAL.h"
#include <Eigen/Dense>

namespace PFClusterCudaHCAL {

  // THE ART OF HARDCODING
  // these numbers should be copied over during initialization
  __constant__ float showerSigma = 10;
  
  __constant__ float recHitEnergyNormEB_1 = 0.1;
  __constant__ float recHitEnergyNormEB_2 = 0.2;
  __constant__ float recHitEnergyNormEB_3 = 0.3;
  __constant__ float recHitEnergyNormEB_4 = 0.3;
  __constant__ float recHitEnergyNormEE_1 = 0.1;
  __constant__ float recHitEnergyNormEE_2_7 = 0.2;
  
  __constant__ float minFracToKeep = 0.0000001;

  __constant__ float seedEThresholdEB_1 = 0.125;
  __constant__ float seedEThresholdEB_2 = 0.25;
  __constant__ float seedEThresholdEB_3 = 0.35;
  __constant__ float seedEThresholdEB_4 = 0.35;
  __constant__ float seedEThresholdEE_1 = 0.1375;
  __constant__ float seedEThresholdEE_2_7 = 0.275;

  __constant__ float seedPt2ThresholdEB = 0.0*0.0;
  __constant__ float seedPt2hresholdEE = 0.0*0.0;

  __constant__ float topoEThresholdEB_1 = 0.1;
  __constant__ float topoEThresholdEB_2 = 0.2;
  __constant__ float topoEThresholdEB_3 = 0.3;
  __constant__ float topoEThresholdEB_4 = 0.3;
  __constant__ float topoEThresholdEE_1 = 0.1;
  __constant__ float topoEThresholdEE_2_7 = 0.2;

  __constant__ int nNeighTopo = 8;
  __constant__ int nNeigh = 4;
  __constant__ int maxSize = 50;
   
 __global__ void seedingKernel_HCAL(
     				    size_t size, 
				    float* pfrh_energy,
				    float* pfrh_pt2,
				    int*   pfrh_isSeed,
				    int*   pfrh_topoId,
				    int*   pfrh_layer,
				    int*   pfrh_depth,
				    int*   neigh4_Ind
				    ) {

   int i = threadIdx.x+blockIdx.x*blockDim.x;

   if(i<size) {        
     if( ( pfrh_layer[i] == 1 && 
	   pfrh_depth[i] == 1 &&
	   pfrh_energy[i]>seedEThresholdEB_1 && 
	   pfrh_pt2[i]>seedPt2ThresholdEB ) ||
	 ( pfrh_layer[i] == 1 && 
	   pfrh_depth[i] == 2 &&
	   pfrh_energy[i]>seedEThresholdEB_2 && 
	   pfrh_pt2[i]>seedPt2ThresholdEB ) ||
	 ( pfrh_layer[i] == 1 && 
	   pfrh_depth[i] == 3 &&
	   pfrh_energy[i]>seedEThresholdEB_3 && 
	   pfrh_pt2[i]>seedPt2ThresholdEB ) ||
	 ( pfrh_layer[i] == 1 && 
	   pfrh_depth[i] == 4 &&
	   pfrh_energy[i]>seedEThresholdEB_4 && 
	   pfrh_pt2[i]>seedPt2ThresholdEB ) ||
	 ( pfrh_layer[i] == 3  && 
	   pfrh_depth[i] == 1  &&
	   pfrh_energy[i]>seedEThresholdEE_1 && 
	   pfrh_pt2[i]>seedPt2hresholdEE)   ||
	 ( pfrh_layer[i] == 3  && 
	   pfrh_depth[i] > 1   &&
	   pfrh_energy[i]>seedEThresholdEE_2_7 && 
	   pfrh_pt2[i]>seedPt2hresholdEE))
       {
	 pfrh_isSeed[i]=1;		 
	 for(int k=0; k<nNeigh; k++){
	   if(neigh4_Ind[nNeigh*i+k]<0) continue; 
	   if(pfrh_energy[i]<pfrh_energy[neigh4_Ind[nNeigh*i+k]]){
	     pfrh_isSeed[i]=0;
	     pfrh_topoId[i]=-1;	     
	     break;
	   }
	 }		
       }
     else{ 
       pfrh_topoId[i]=-1;
       pfrh_isSeed[i]=0;
       	    
     }     
   }
 }
  
  __global__ void topoKernel_HCAL( 
				  size_t size,
				  float* pfrh_energy,
				  int* pfrh_topoId,
				  int* pfrh_layer,
				  int* pfrh_depth,
				  int* neigh8_Ind
				  ) {
    
    int l = threadIdx.x+blockIdx.x*blockDim.x;
	if(l<size) {
	  //printf("layer: %d",pfrh_layer[l]);
	  for(int k=0; k<nNeighTopo; k++){
	    if( neigh8_Ind[nNeighTopo*l+k] > -1 && 
		pfrh_topoId[l] < pfrh_topoId[neigh8_Ind[nNeighTopo*l+k]] && 
		( (pfrh_layer[l] == 3 &&  pfrh_depth[l] == 1 && pfrh_energy[l]>topoEThresholdEE_1)   ||
		  (pfrh_layer[l] == 3 &&  pfrh_depth[l] >  1 && pfrh_energy[l]>topoEThresholdEE_2_7) ||
		  (pfrh_layer[l] == 1 &&  pfrh_depth[l] == 1 && pfrh_energy[l]>topoEThresholdEB_1) ||
		  (pfrh_layer[l] == 1 &&  pfrh_depth[l] == 2 && pfrh_energy[l]>topoEThresholdEB_2) ||
		  (pfrh_layer[l] == 1 &&  pfrh_depth[l] == 3 && pfrh_energy[l]>topoEThresholdEB_3) ||
		  (pfrh_layer[l] == 1 &&  pfrh_depth[l] == 4 && pfrh_energy[l]>topoEThresholdEB_4)
		  ) 
		)
	      {
		pfrh_topoId[l]=pfrh_topoId[neigh8_Ind[nNeighTopo*l+k]];
	      }
	  }				       
	}//loop end
  }
  
  
  __global__ void pfClusterKernel_HCAL_step1(
					      
					     size_t size,
					     float* pfrh_x,
					     float* pfrh_y,
					     float* pfrh_z,
					     float* pfrh_energy,
					     int* pfrh_topoId,
					     int* pfrh_isSeed,
					     int* pfrh_layer,
					     int* pfrh_depth,
					     
					     float* pfrhfrac, 
					     int* pfrhfracind
					     ) {
    
    int l = threadIdx.x+blockIdx.x*blockDim.x;
    if(l<size) {
      
      int countFracPerRh = 0;
      float fracTot = 0.;
      if(pfrh_isSeed[l] == 1){//<-- if it is seed, create first entry in pfcl/frac and fr
	
	pfrhfrac[l*maxSize+countFracPerRh] = 1;
	pfrhfracind[l*maxSize+countFracPerRh] = l;
	fracTot = fracTot+1;
	countFracPerRh++;


	for(int p=0; p<size; p++){
	  
	  if(pfrh_topoId[l] == pfrh_topoId[p] && pfrh_topoId[p]>0. && pfrh_isSeed[p] != 1){ //<-- only if rechits are in the same topocluster they should be part of a pfcluster, if current rechit is seed we record those which are not seeds		
	    //measure distance 
	    float dist2 = 
	       (pfrh_x[l] - pfrh_x[p])*(pfrh_x[l] - pfrh_x[p])
	      +(pfrh_y[l] - pfrh_y[p])*(pfrh_y[l] - pfrh_y[p])
	      +(pfrh_z[l] - pfrh_z[p])*(pfrh_z[l] - pfrh_z[p]);
	    
	    float d2 = dist2 / (showerSigma*showerSigma);	  
	    //if(d2>100.) printf("the distance in units of showerSigma is larger than 100...");
		
	    float fraction = -1.;
	    
	    if(pfrh_layer[p] == 1 && pfrh_depth[p] == 1) { fraction = pfrh_energy[p] / recHitEnergyNormEB_1 * expf(-0.5 * d2); }
	    if(pfrh_layer[p] == 1 && pfrh_depth[p] == 2) { fraction = pfrh_energy[p] / recHitEnergyNormEB_2 * expf(-0.5 * d2); }
	    if(pfrh_layer[p] == 1 && pfrh_depth[p] == 3) { fraction = pfrh_energy[p] / recHitEnergyNormEB_3 * expf(-0.5 * d2); }
	    if(pfrh_layer[p] == 1 && pfrh_depth[p] == 4) { fraction = pfrh_energy[p] / recHitEnergyNormEB_4 * expf(-0.5 * d2); }
	    if(pfrh_layer[p] == 3 && pfrh_depth[p] == 1) { fraction = pfrh_energy[p] / recHitEnergyNormEE_1 * expf(-0.5 * d2); }
	    if(pfrh_layer[p] == 3 && pfrh_depth[p] > 1 ) { fraction = pfrh_energy[p] / recHitEnergyNormEE_2_7 * expf(-0.5 * d2); }

	    if(fraction==-1.) printf("FRACTION is NEGATIVE!!!");
	    
	    if(d2 < 100 /*&&  fraction > minFracToKeep*/){ 
	      pfrhfracind[l*maxSize+countFracPerRh] = p;
	      countFracPerRh++;
	    }//end of putting Seed to list of Seeds per rechit
	  }// if rechit in neighbourhood is seed
	}//<== loop of rechits	    
      }//<== if it is seed
	
      
      
      if(pfrh_isSeed[l] != 1){ //<-- seeds are not part of other clusters
	
	//loop over rechits to find seeds the rechit could be part of the cluster 
	for(int p=0; p<size; p++){
	  
	  if(pfrh_topoId[l] == pfrh_topoId[p] && pfrh_topoId[p]>0. && pfrh_isSeed[p] == 1){ //<-- only if rechits are in the same topocluster they should be part of a pfcluster, if current rechit is not seed we record those which are seeds		
	    //measure distance to seed
	    float dist2 = 
	       (pfrh_x[l] - pfrh_x[p])*(pfrh_x[l] - pfrh_x[p])
	      +(pfrh_y[l] - pfrh_y[p])*(pfrh_y[l] - pfrh_y[p])
	      +(pfrh_z[l] - pfrh_z[p])*(pfrh_z[l] - pfrh_z[p]);
	    
	    float d2 = dist2 / (showerSigma*showerSigma);	  
	    //if(d2>100.) printf("the distance in units of showerSigma is larger than 100...");
		
	    float fraction = -1.;
	    if(pfrh_layer[p] == 1 && pfrh_depth[p] == 1) { fraction = pfrh_energy[p] / recHitEnergyNormEB_1 * expf(-0.5 * d2); }
	    if(pfrh_layer[p] == 1 && pfrh_depth[p] == 2) { fraction = pfrh_energy[p] / recHitEnergyNormEB_2 * expf(-0.5 * d2); }
	    if(pfrh_layer[p] == 1 && pfrh_depth[p] == 3) { fraction = pfrh_energy[p] / recHitEnergyNormEB_3 * expf(-0.5 * d2); }
	    if(pfrh_layer[p] == 1 && pfrh_depth[p] == 4) { fraction = pfrh_energy[p] / recHitEnergyNormEB_4 * expf(-0.5 * d2); }
	    if(pfrh_layer[p] == 3 && pfrh_depth[p] == 1) { fraction = pfrh_energy[p] / recHitEnergyNormEE_1 * expf(-0.5 * d2); }
	    if(pfrh_layer[p] == 3 && pfrh_depth[p] > 1 ) { fraction = pfrh_energy[p] / recHitEnergyNormEE_2_7 * expf(-0.5 * d2); }
	    
	    if(fraction==-1.) printf("FRACTION is NEGATIVE!!!");
	    
	    if(d2 < 100 /*&&  fraction > minFracToKeep*/){ 
	      pfrhfrac[l*maxSize+countFracPerRh] = fraction;
	      pfrhfracind[l*maxSize+countFracPerRh] = p;
	      fracTot = fracTot + fraction;
	      countFracPerRh++;
	    }//end of putting Seed to list of Seeds per rechit
	  }// if rechit in neighbourhood is seed
	}//<== loop of rechits	    
      }//<== if it is not seed
      
      //normalize the fractions
      for(int m = 0; m<countFracPerRh; m++){
	pfrhfrac[l*maxSize+m] = pfrhfrac[l*maxSize+m]/(fracTot); 
      }//<== end normalise fractions
    }//<== if l<size 
  }//<== end of function
  
 

__global__ void pfClusterKernel_HCAL_step2_V2(					     
					     size_t size, 
					     int* pfrh_isSeed,
					     float* pfrh_energy,
					      
					     float* pfrhfrac, 
					     int* pfrhfracind, 
					     int* pcrhfracind,
					     float* pcrhfrac
					     ) {
    
    int l = threadIdx.x+blockIdx.x*blockDim.x;
    if(l<size) {
      
      int nFracPerSeed=0;

      if(pfrh_isSeed[l]==1)
	{
	  pcrhfracind[l*maxSize] = l;
	  pcrhfrac[l*maxSize] = 1;
	  nFracPerSeed++;
	for(int i=1; i<maxSize; i++)
	  {
	  if(pfrhfracind[l*maxSize+i] > -1)
	    {
	      for(int j=0; j<maxSize; j++)
		{
		  if(pfrhfracind[ pfrhfracind[l*maxSize+i]*maxSize + j ] == l && pfrhfrac[pfrhfracind[l*maxSize+i]*maxSize + j]>minFracToKeep)
		    {
		      
		      pcrhfracind[l*maxSize+nFracPerSeed]=pfrhfracind[l*maxSize+i];
		      pcrhfrac[l*maxSize+nFracPerSeed]=pfrhfrac[pfrhfracind[l*maxSize+i]*maxSize + j ];
		      nFracPerSeed++;
		      break;
		    }
		}
	    }
	  if(pfrhfracind[l*maxSize+i] < 0) break;
	}
	
      }

    }//end of l<size
  }//end of function
   
  void PFRechitToPFCluster_HCAL(size_t size, 
				float* pfrh_x, 
				float* pfrh_y, 
				float* pfrh_z, 
				float* pfrh_energy, 
				float* pfrh_pt2, 				 				
				int* pfrh_isSeed,
				int* pfrh_topoId, 
				int* pfrh_layer, 
				int* pfrh_depth, 
				int* neigh8_Ind, 
				int* neigh4_Ind, 
				
				float* pfrhfrac, 
				int* pfrhfracind, 
				int* pcrhfracind,
				float* pcrhfrac
				)
  { 
    //seeding
    if(size>0) seedingKernel_HCAL<<<(size+512-1)/512, 512>>>( size,  pfrh_energy,   pfrh_pt2,   pfrh_isSeed,  pfrh_topoId,  pfrh_layer,pfrh_depth,  neigh4_Ind);
    //hipDeviceSynchronize();
    
    //topoclustering
    for(int j=0;j<16;j++){
      if(size>0) topoKernel_HCAL<<<(size+512-1)/512, 512>>>( size, pfrh_energy,  pfrh_topoId,  pfrh_layer, pfrh_depth, neigh8_Ind);	    
      //hipDeviceSynchronize();
    }
    
    //pfclustering
    if(size>0) pfClusterKernel_HCAL_step1<<<(size+512-1)/512, 512>>>( size, pfrh_x,  pfrh_y,  pfrh_z,  pfrh_energy, pfrh_topoId,  pfrh_isSeed,  pfrh_layer, pfrh_depth, pfrhfrac, pfrhfracind);
    //hipDeviceSynchronize();
    
    if(size>0) pfClusterKernel_HCAL_step2_V2<<<(size+512-1)/512, 512>>>(size, pfrh_isSeed, pfrh_energy, pfrhfrac, pfrhfracind, pcrhfracind, pcrhfrac);
    hipDeviceSynchronize();
    
    cudaCheck(hipGetLastError());	  
  }
}  // namespace cudavectors
